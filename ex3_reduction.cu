
#include <hip/hip_runtime.h>
#include <iostream>

#define CUDA_CHECK(code) { cuda_check((code), __FILE__, __LINE__); }
inline void cuda_check(hipError_t code, const char *file, int line) {
    if(code != hipSuccess) {
        fprintf(stderr,"%s:%d: [CUDA ERROR] %s\n", file, line, hipGetErrorString(code));
    }
}

constexpr auto block_dim = 256;  // constexpr equivalent to blockDim.x in CUDA kernel
constexpr auto block_count = 256; // constexpr equivalent to gridDim.x in CUDA kernel


int main(int argc, char const *argv[])
{
    const int N = argc >= 2 ? std::stoi(argv[1]) : 1e6;
    std::cout << "N = " << N << std::endl;

    float *x, *y;

    float host_expected_result = 0;
    float device_result = 0;

    x = (float*)malloc(N * sizeof(float));
    y = (float*)malloc(N * sizeof(float));

    for (int i = 0; i < N; i++) {
        x[i] = 2 * float(std::rand()) / RAND_MAX - 1; // random float in (-1,+1)
        y[i] = 2 * float(std::rand()) / RAND_MAX - 1; // random float in (-1,+1)
        host_expected_result += x[i] * y[i];
    }

    // ...

    std::cout << "host_expected_result = " << host_expected_result << std::endl;
    std::cout << "device_result = " << device_result << std::endl;

    free(x);
    free(y);
    
    return 0;
}