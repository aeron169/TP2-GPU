#include "hip/hip_runtime.h"
#include <string>
#include <iostream>
#include "image.h"

#define CUDA_CHECK(code) { cuda_check((code), __FILE__, __LINE__); }
inline void cuda_check(hipError_t code, const char *file, int line) {
    if(code != hipSuccess) {
        fprintf(stderr,"%s:%d: [CUDA ERROR] %s\n", file, line, hipGetErrorString(code));
    }
}


template <typename T>
__device__ inline T* get_ptr(T *img, int i, int j, int C, size_t pitch) {
    // ...
	return 0;
}


__global__
void process(int N, int M, int C, int pitch, float* img)
{
    // ...
}


int main(int argc, char const *argv[])
{
    const std::string filename = argc >= 2 ? argv[1] : "image.jpg";
    std::cout << "filename = " << filename << std::endl;

    int M = 0;
    int N = 0;
    int C = 0;
    float* img = image::load(filename, &N, &M, &C);
    std::cout << "N (columns, width) = " << N << std::endl;
    std::cout << "M (rows, height) = " << M << std::endl;
    std::cout << "C (channels, depth) = " << C << std::endl;

    // ...

    image::save("result.jpg", N, M, C, img);

    free(img);

    return 0;
}
