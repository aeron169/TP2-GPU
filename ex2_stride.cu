
#include <hip/hip_runtime.h>
#include <iostream>

#define CUDA_CHECK(code) { cuda_check((code), __FILE__, __LINE__); }
inline void cuda_check(hipError_t code, const char *file, int line) {
    if(code != hipSuccess) {
        fprintf(stderr,"%s:%d: [CUDA ERROR] %s\n", file, line, hipGetErrorString(code));
    }
}

int main(int argc, char const *argv[])
{
    const int N = argc >= 2 ? std::stoi(argv[1]) : 1e6;
    std::cout << "N = " << N << std::endl;

    float *x, *y;
    x = (float*)malloc(N * sizeof(float));
    y = (float*)malloc(N * sizeof(float));

    for (int i = 0; i < N; i++) {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    // ...

    free(x);
    free(y);

    return 0;
}